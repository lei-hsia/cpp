#include "hip/hip_runtime.h"
/*
    第一个GPU程序:
    0,1,...63：前64个自然数求平方和
    只写最核心的部分
*/

// CPU code: sqaure each element of an array
// 只有一个线程, one independent path of execution through code;
// no explicit parallelism;
// 64次乘，如果一次 2 nanoseconds, 一共128纳秒
for (int i=0; i<64; ++i) {
    out[i] = in[i] * in[i];
}

// GPU:
// GPU 只是表述了kernel的任务, i.e. Express out = in * in; 并没有
// 写任何有关parallel的东西;
//
// CPU:
// 1. allocate memory
// 2. copy data to/from GPU
// 3. launch kernel <---- specifies degree of parallelism：这一步才是实现并发的地方!!!
//
// CPU code: squareKernel<<<64>>>(outArray, inArray)
//           这里做的是,写一个kernel这个kernel的名字叫squareKernel;
//           显式声明64个这样的kernel instances, 每个instance都会
//           执行一次GPU的kernel中的任务;
//     launch 64 instances of same program:
//          每个instance都知道自己是在哪个thread;
//     ---> assign thread N 解决第 N 个array;
//
//
// 1. 自己写kernel, 然后kernel每次在一个thread上面运行;
//     kernel 实际上就是要运行的program;
// 2. CPU launch很多threads, 每个thread都自己独立运行那个kernel
//
// 64次乘, 每个都是花费10纳秒: 因为并行所以一共也是10纳秒;
// GPU的latency是比CPU的时间长的，但是因为可以并行所以还是10纳秒

#include <stdio.h>
// kernel: 实际上就是要run的函数, 在GPU上运行
__global__ void square(float * d_out, float * d_in) {
    // kernel中, threadIdx是CUDA内置的变量, 这是一个C的struct,通过这个每个线程知道了自己的index
    // threadIdx有3个属性x,y,z;这里都用x以示区分
    // 所以main函数中创建了64个instances, 那么kernel函数中.x就是从0,1,...63
    int idx = threadIdx.x;
    float f = d_in[idx];  // read element: 从全局memory中读取这个index对应的元素值存进f
    d_out[idx] = f * f;
}

int main(int argc, char ** argv) {
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // h: host;    d: device

    // generate input array on the host
    float h_in[ARRAY_SIZE];
    for (int i=0; i<ARRAY_SIZE; ++i) {
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    // declare GPU memory pointers
    float *d_in;
    float *d_out;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    // transfer array to GPU (transfer input)
    // hipMemcpy(dst, src, t_count)
    hipMemcpy(d_in, h_in, ARRAY_BYTES,  hipMemcpyHostToDevice);

    // launch the kernel (实际运行)
    // 函数定义为 __global__ void Func(float* parameter);
    // 这样被调用: Func<<<Dg, Db, Ns>>>(parameter);
    // Dg: dimension and size of grid;
    // Db: dimension and size of block;
    // Ns: shared memory中除静态内存外, 动态分配给每个block的内存
    // S: hipStream_t的数量, default为0
    square<<<1, ARRAY_SIZE>>>(d_out, d_in);  // kernel传递的是kernel函数的参数/GPU程序的参数

    // copy back the result
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // print out the resulting array
    for (int i=0; i<ARRAY_SIZE; ++i) {
        printf("%f", h_out[i]);
        printf(((i%4) != 3) ? "\t" : "\n");
    }

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
