/* random number generator in CUDA
	
	Brownian motion evolves w. Normally distributed
	random steps, I use normal generator.
*/
#include <hiprand.h>

// var declaration
hiprandGenerator_t curandGenerator;

// hiprandCreateGenerator(generator_var, generator_method)
// use Mersenne Twister algorithm
hiprandCreateGenerator(&curandGenerator, HIPRAND_RNG_PSEUDO_MTGP32);

// set seed
hiprandSetPseudoRandomGeneratorSeed(curandGenerator, 1234ULL);

// generate normally distributed random numbers
hiprandGenerateNormal(curandGenerator, d_normal.getData(),
					N_NORMALS, 0.0f, sqrdt);

// destroy generator
hiprandDestroyGenerator(curandGenerator);