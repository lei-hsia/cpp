#include "hip/hip_runtime.h"
#include "kernel.h"

// GPU Monte Carlo simulation
__global__ void mc_kernel(
  float * d_s,
  float T,
  float B,
  float K,
  float S0,
  float sigma,
  float mu,
  float r,
  float dt,
  float * d_normals,
  unsigned N_STEPS,
  unsigned N_PATHS) {
      const unsigned tid = threadIdx.x;
      const unsigned bid = blockIdx.x;
      const unsigned bsz = blockDim.x;
      int s_idx = tid + bid * bsz;  // price array
      int n_idx = tid + bid * bsz;  // random num array
      float s_curr = S0;
      // array index is lower than max number of threads
      if (s_idx < N_PATHS) {
          int n = 0;
          // when reaches max steps or hit barrier, no need to loop
          do {
              s_curr = s_curr + mu*s_curr*dt + sigma*s_curr*d_normals[n_idx];
              ++n_idx;
              ++n;
          }
          while (n < N_STEPS && s_curr > B);
          // end loop, calculate payoff
          double payoff = (s_curr>K ? s_curr-K : 0.0);
          _syncthreads();
          // save to device price array
          d_s[s_idx] = exp(-r*T) * payoff;
      }
  }
// mc_call wrapper: call real kernel function
void mc_call(
    float * d_s,
    float T,
    float B, float K,
    float S0, float sigma, float mu, float r,

    float dt, float * d_normals,

    unsigned N_STEPS, unsigned N_PATHS) {
        const unsigned BLOCK_SIZE = 1024;
        const unsigned GRID_SIZE = ceil(float(N_PATHS)/float(BLOCK_SIZE));
        mc_kernel<<<GRID_SIZE, BLOCK_SIZE>>> (
            d_s,T,B,K,S0,sigma,mu,r,dt,d_normals,N_STEPS,N_PATHS
        );
    }
